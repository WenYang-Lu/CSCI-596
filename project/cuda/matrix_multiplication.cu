// Using CUDA device to calculate pi
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdlib.h>

__global__ void cal_matrix_multiplication(int* A, int* B, int* C, int m, int n, int p) {
	int r = threadIdx.x;
	int c = threadIdx.y;
	int i;
	for (i = 0; i < n; i++) {
		C[r*p + c] += A[r*n + i] * B[i*p + c];
	}
}

void func(int m, int n, int p) {
	int i;
	float cpu1,cpu2;
	int* A = (int*)malloc(m * n * sizeof(int));
	int* B = (int*)malloc(n * p * sizeof(int));

	for (i = 0; i < m * n; i++)
        A[i] = i;
	for (i = 0; i < n * p; i++)
        B[i] = i;
	
	cpu1 = ((double) clock())/CLOCKS_PER_SEC;

	dim3 dimGrid(1,1,1);  // Grid dimensions
	dim3 dimBlock(m,p,1);  // Block dimensions
	int* C = (int*)malloc(m * p * sizeof(int)); //  Allocate array on host
	int *A_dev, *B_dev, *C_dev; // Pointer to device arrays
	hipMalloc((void **) &A_dev, m*n*sizeof(int));  // Allocate array on device
	hipMalloc((void **) &B_dev, n*p*sizeof(int));  // Allocate array on device
	hipMalloc((void **) &C_dev, m*p*sizeof(int));  // Allocate array on device
	hipMemcpy(A_dev, A, m*n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(B_dev, B, n*p*sizeof(int), hipMemcpyHostToDevice);
	// Initialize array in device to 0
	hipMemset(C_dev, 0, m*p*sizeof(int));
	// Do calculation on device
	cal_matrix_multiplication <<<dimGrid, dimBlock>>> (A_dev, B_dev, C_dev, m, n, p); // call CUDA kernel
	// Retrieve result from device and store it in host array
	hipMemcpy(C, C_dev, m*p*sizeof(int), hipMemcpyDeviceToHost);

	cpu2 = ((double) clock())/CLOCKS_PER_SEC;
  	printf("%d, Execution time (s) = %le\n",m, cpu2-cpu1);
	// Print results
	/*
	printf("A = \n");
	for (i = 0; i < m; i++) {
		for (j = 0; j < n; j++) {
        	printf("%d ", A[i*n+j]);
		}
		printf("\n");
	}
	printf("B = \n");
	for (i = 0; i < n; i++) {
		for (j = 0; j < p; j++) {
        	printf("%d ", B[i*p+j]);
		}
		printf("\n");
	}
	printf("C = \n");
	for (i = 0; i < m; i++) {
		for (j = 0; j < p; j++) {
        	printf("%d ", C[i*p+j]);
		}
		printf("\n");
	}
	*/

	// Cleanup
	free(A);
	free(B);
	free(C);

	hipFree(A_dev);
	hipFree(B_dev);
	hipFree(C_dev);
}
// Main routine that executes on the host
int main(void) {
	int i;
	for (i = 1; i < 1025; i*=2)
		func(i,i,i);

	return 0;
}